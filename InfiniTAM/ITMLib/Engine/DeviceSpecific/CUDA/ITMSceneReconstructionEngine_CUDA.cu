#include "hip/hip_runtime.h"
// Copyright 2014-2015 Isis Innovation Limited and the authors of InfiniTAM

#include "ITMSceneReconstructionEngine_CUDA.h"
#include "ITMCUDAUtils.h"
#include "../../DeviceAgnostic/ITMSceneReconstructionEngine.h"
#include "../../../Objects/ITMRenderState_VH.h"
#include "../../../ITMLib.h"

struct AllocationTempData {
	int noAllocatedVoxelEntries;
	int noAllocatedExcessEntries;
	int noVisibleEntries;
};

using namespace ITMLib::Engine;

template<class TVoxel, bool stopMaxW, bool approximateIntegration>
__global__ void integrateIntoScene_device(TVoxel *localVBA, const ITMHashEntry *hashTable, int *noVisibleEntryIDs,
	const Vector4u *rgb, Vector2i rgbImgSize, const float *depth, Vector2i imgSize, Matrix4f M_d, Matrix4f M_rgb, Vector4f projParams_d, 
	Vector4f projParams_rgb, float _voxelSize, float mu, int maxW);

template<class TVoxel, bool stopMaxW, bool approximateIntegration>
__global__ void integrateIntoScene_device(TVoxel *voxelArray, const ITMPlainVoxelArray::ITMVoxelArrayInfo *arrayInfo,
	const Vector4u *rgb, Vector2i rgbImgSize, const float *depth, Vector2i depthImgSize, Matrix4f M_d, Matrix4f M_rgb, Vector4f projParams_d, 
	Vector4f projParams_rgb, float _voxelSize, float mu, int maxW);

__global__ void buildHashAllocAndVisibleType_device(uchar *entriesAllocType, uchar *entriesVisibleType, Vector4s *blockCoords, const float *depth,
	Matrix4f invM_d, Vector4f projParams_d, float mu, Vector2i _imgSize, float _voxelSize, ITMHashEntry *hashTable, float viewFrustum_min,
	float viewFrustrum_max, int *locks);

__global__ void allocateVoxelBlocksList_device(int *voxelAllocationList,
											   int *excessAllocationList,
											   ITMHashEntry *hashTable,
											   int noTotalEntries,
											   AllocationTempData *allocData,
											   uchar *entriesAllocType,
											   uchar *entriesVisibleType,
											   Vector4s *blockCoords,
											   int currentFrame);

__global__ void reAllocateSwappedOutVoxelBlocks_device(int *voxelAllocationList, ITMHashEntry *hashTable, int noTotalEntries,
	AllocationTempData *allocData, uchar *entriesVisibleType);

__global__ void setToType3(uchar *entriesVisibleType, int *visibleEntryIDs, int noVisibleEntries, ITMHashEntry *hashTable);

template<bool useSwapping>
__global__ void buildVisibleList_device(ITMHashEntry *hashTable, ITMHashSwapState *swapStates, int noTotalEntries,
	int *visibleEntryIDs, AllocationTempData *allocData, uchar *entriesVisibleType,
	Matrix4f M_d, Vector4f projParams_d, Vector2i depthImgSize, float voxelSize);

// TODO(andrei): Redo documentation after you finish implementing this.
/// \brief Erases blocks whose weight is smaller than 'maxWeight', and marks blocks which become
///        empty in the process as pending deallocation in `outBlocksToDeallocate`.
/// \tparam TVoxel The type of voxel representation to operate on (grayscale/color, float/short, etc.)
/// \param localVBA The raw storage where the hash map entries reside.
/// \param hashTable Maps entry IDs to addresses in the local VBA.
/// \param visibleEntryIDs A list of blocks on which to operate (typically, this is the list
///                        containing the visible blocks $k$ frames ago. The size of the list should
///                        be known in advance, and be implicitly range-checked by setting the grid
///                        size's x dimension to it.
template<class TVoxel>
__global__ void decay_device(TVoxel *localVBA,
							 ITMHashEntry *hashTable,
							 int *visibleEntryIDs,
							 int minAge,
							 int maxWeight,
							 int *voxelAllocationList,
							 int *lastFreeBlockId,
							 int *locks,
							 int currentFrame
);

/// \brief Used to perform voxel decay on all voxels in a volume.
template<class TVoxel>
__global__ void decayFull_device(
		const Vector4s *visibleBlockGlobalPos,
		TVoxel *localVBA,
		ITMHashEntry *hashTable,
		int maxWeight,
		int *lastFreeBlockId,
		int *voxelAllocationList);


// host methods

template<class TVoxel>
ITMSceneReconstructionEngine_CUDA<TVoxel,ITMVoxelBlockHash>::ITMSceneReconstructionEngine_CUDA(void) 
{
	ITMSafeCall(hipMalloc((void**)&allocationTempData_device, sizeof(AllocationTempData)));
	ITMSafeCall(hipHostMalloc((void**)&allocationTempData_host, sizeof(AllocationTempData)));

	int noTotalEntries = ITMVoxelBlockHash::noTotalEntries;
	ITMSafeCall(hipMalloc((void**)&entriesAllocType_device, noTotalEntries));
	ITMSafeCall(hipMalloc((void**)&blockCoords_device, noTotalEntries * sizeof(Vector4s)));

	ITMSafeCall(hipMalloc((void**)&blocksToDeallocate_device, maxBlocksToDeallocate * sizeof(int)));
	ITMSafeCall(hipMalloc((void**)&blocksToDeallocateCount_device, 1 * sizeof(int)));
	ITMSafeCall(hipMalloc((void**)&lastFreeBlockId_device, 1 * sizeof(int)));
	ITMSafeCall(hipMalloc(&locks_device, SDF_BUCKET_NUM * sizeof(int)));
}

template<class TVoxel>
ITMSceneReconstructionEngine_CUDA<TVoxel,ITMVoxelBlockHash>::~ITMSceneReconstructionEngine_CUDA(void) 
{
	ITMSafeCall(hipHostFree(allocationTempData_host));
	ITMSafeCall(hipFree(allocationTempData_device));
	ITMSafeCall(hipFree(entriesAllocType_device));
	ITMSafeCall(hipFree(blockCoords_device));

	ITMSafeCall(hipFree(blocksToDeallocate_device));
	ITMSafeCall(hipFree(blocksToDeallocateCount_device));
	ITMSafeCall(hipFree(lastFreeBlockId_device));
	ITMSafeCall(hipFree(locks_device));
}

template<class TVoxel>
void ITMSceneReconstructionEngine_CUDA<TVoxel,ITMVoxelBlockHash>::ResetScene(ITMScene<TVoxel, ITMVoxelBlockHash> *scene)
{
	int numBlocks = scene->index.getNumAllocatedVoxelBlocks();
	int blockSize = scene->index.getVoxelBlockSize();

	totalDecayedBlockCount = 0;
	// Clean up the visible frame queue used in voxel decay.
	while (! frameVisibleBlocks.empty()) {
		delete frameVisibleBlocks.front().blockIDs;
		frameVisibleBlocks.pop();
	}

	TVoxel *voxelBlocks_ptr = scene->localVBA.GetVoxelBlocks();
	memsetKernel<TVoxel>(voxelBlocks_ptr, TVoxel(), numBlocks * blockSize);
	int *vbaAllocationList_ptr = scene->localVBA.GetAllocationList();
	fillArrayKernel<int>(vbaAllocationList_ptr, numBlocks);
	scene->localVBA.lastFreeBlockId = numBlocks - 1;

	ITMHashEntry tmpEntry;
	memset(&tmpEntry, 0, sizeof(ITMHashEntry));
	tmpEntry.ptr = -2;
	ITMHashEntry *hashEntry_ptr = scene->index.GetEntries();
	memsetKernel<ITMHashEntry>(hashEntry_ptr, tmpEntry, scene->index.noTotalEntries);
	int *excessList_ptr = scene->index.GetExcessAllocationList();
	fillArrayKernel<int>(excessList_ptr, SDF_EXCESS_LIST_SIZE);

	scene->index.SetLastFreeExcessListId(SDF_EXCESS_LIST_SIZE - 1);
}

template<class TVoxel>
void ITMSceneReconstructionEngine_CUDA<TVoxel, ITMVoxelBlockHash>::AllocateSceneFromDepth(
		ITMScene<TVoxel, ITMVoxelBlockHash> *scene,
		const ITMView *view,
		const ITMTrackingState *trackingState,
		const ITMRenderState *renderState,
		bool onlyUpdateVisibleList
) {
	Vector2i depthImgSize = view->depth->noDims;
	float voxelSize = scene->sceneParams->voxelSize;

	Matrix4f M_d, invM_d;
	Vector4f projParams_d, invProjParams_d;

	ITMRenderState_VH *renderState_vh = (ITMRenderState_VH*)renderState;
	M_d = trackingState->pose_d->GetM(); M_d.inv(invM_d);

	projParams_d = view->calib->intrinsics_d.projectionParamsSimple.all;
	invProjParams_d = projParams_d;
	invProjParams_d.x = 1.0f / invProjParams_d.x;
	invProjParams_d.y = 1.0f / invProjParams_d.y;

	float mu = scene->sceneParams->mu;

	float *depth = view->depth->GetData(MEMORYDEVICE_CUDA);
	int *voxelAllocationList = scene->localVBA.GetAllocationList();
	int *excessAllocationList = scene->index.GetExcessAllocationList();
	ITMHashEntry *hashTable = scene->index.GetEntries();
	ITMHashSwapState *swapStates = scene->useSwapping ? scene->globalCache->GetSwapStates(true) : 0;

	// The sum of the nr of buckets, plus the excess list size
	int noTotalEntries = scene->index.noTotalEntries;
	int *visibleEntryIDs = renderState_vh->GetVisibleEntryIDs();
	uchar *entriesVisibleType = renderState_vh->GetEntriesVisibleType();

	dim3 cudaBlockSizeHV(16, 16);
	dim3 gridSizeHV((int)ceil((float)depthImgSize.x / (float)cudaBlockSizeHV.x), (int)ceil((float)depthImgSize.y / (float)cudaBlockSizeHV.y));

	dim3 cudaBlockSizeAL(256, 1);
	dim3 gridSizeAL((int)ceil((float)noTotalEntries / (float)cudaBlockSizeAL.x));

	dim3 cudaBlockSizeVS(256, 1);
	dim3 gridSizeVS((int)ceil((float)renderState_vh->noVisibleEntries / (float)cudaBlockSizeVS.x));

	float oneOverVoxelSize = 1.0f / (voxelSize * SDF_BLOCK_SIZE);

	AllocationTempData *tempData = static_cast<AllocationTempData*>(allocationTempData_host);
	tempData->noAllocatedVoxelEntries = scene->localVBA.lastFreeBlockId;
	tempData->noAllocatedExcessEntries = scene->index.GetLastFreeExcessListId();
	tempData->noVisibleEntries = 0;
	ITMSafeCall(hipMemcpyAsync(allocationTempData_device, tempData, sizeof(AllocationTempData), hipMemcpyHostToDevice));

	ITMSafeCall(hipMemsetAsync(entriesAllocType_device, 0, sizeof(unsigned char)* noTotalEntries));

	if (gridSizeVS.x > 0) {
		// Flags all previously visible blocks accordingly (runs for every element in the
		// visibleEntryIDs list).
		// 0 = invisible (I think)
		// 1 = visible and in memory
		// 2 = visible but swapped out
		// 3 = visible at previous frame and in memory
		setToType3<<<gridSizeVS, cudaBlockSizeVS>>>(
				entriesVisibleType,
				visibleEntryIDs,
				renderState_vh->noVisibleEntries,
				hashTable);
	}

	// TODO(andrei): If locking is useful, do it properly...
	int *locks_device;
	ITMSafeCall(hipMalloc(&locks_device, sizeof(int) * SDF_BUCKET_NUM));
	ITMSafeCall(hipMemset( locks_device, 0, sizeof(int) * SDF_BUCKET_NUM));

	buildHashAllocAndVisibleType_device << <gridSizeHV, cudaBlockSizeHV >> >(entriesAllocType_device, entriesVisibleType,
		blockCoords_device, depth, invM_d, invProjParams_d, mu, depthImgSize, oneOverVoxelSize, hashTable,
		scene->sceneParams->viewFrustum_min, scene->sceneParams->viewFrustum_max, locks_device);

	ITMSafeCall(hipFree(locks_device));

	bool useSwapping = scene->useSwapping;
	if (onlyUpdateVisibleList) useSwapping = false;
	if (!onlyUpdateVisibleList)
	{
		allocateVoxelBlocksList_device<<<gridSizeAL, cudaBlockSizeAL>>>(
				voxelAllocationList,
				excessAllocationList, hashTable,
				noTotalEntries,
				(AllocationTempData *) allocationTempData_device,
				entriesAllocType_device,
				entriesVisibleType,
				blockCoords_device,
				frameIdx);
	}

	if (useSwapping) {
		buildVisibleList_device<true> << <gridSizeAL, cudaBlockSizeAL >> >(hashTable, swapStates, noTotalEntries, visibleEntryIDs,
			(AllocationTempData*)allocationTempData_device, entriesVisibleType, M_d, projParams_d, depthImgSize, voxelSize);
	}
	else {
		buildVisibleList_device<false> << <gridSizeAL, cudaBlockSizeAL >> >(hashTable, swapStates, noTotalEntries, visibleEntryIDs,
			(AllocationTempData*)allocationTempData_device, entriesVisibleType, M_d, projParams_d, depthImgSize, voxelSize);
	}

	if (useSwapping)
	{
		reAllocateSwappedOutVoxelBlocks_device << <gridSizeAL, cudaBlockSizeAL >> >(voxelAllocationList, hashTable, noTotalEntries,
			(AllocationTempData*)allocationTempData_device, entriesVisibleType);
	}

	ITMSafeCall(hipMemcpy(tempData, allocationTempData_device, sizeof(AllocationTempData), hipMemcpyDeviceToHost));
	renderState_vh->noVisibleEntries = tempData->noVisibleEntries;
	scene->localVBA.lastFreeBlockId = tempData->noAllocatedVoxelEntries;
	scene->index.SetLastFreeExcessListId(tempData->noAllocatedExcessEntries);

	// visibleEntryIDs is now populated with block IDs which are visible.
	int totalBlockCount = scene->index.getNumAllocatedVoxelBlocks();
	size_t visibleBlockCount = static_cast<size_t>(tempData->noVisibleEntries);

	size_t visibleEntryIDsByteCount = visibleBlockCount * sizeof(int);
	auto *visibleEntryIDsCopy = new ORUtils::MemoryBlock<int>(
			visibleEntryIDsByteCount, MEMORYDEVICE_CUDA);

	if (visibleEntryIDsByteCount > 0) {
		ITMSafeCall(hipMemcpy(visibleEntryIDsCopy->GetData(MEMORYDEVICE_CUDA),
							   visibleEntryIDs,
							   visibleEntryIDsByteCount,
							   hipMemcpyDeviceToDevice));
	}
	VisibleBlockInfo visibleBlockInfo = {
		visibleBlockCount,
		frameIdx,
		visibleEntryIDsCopy,
	};
	frameIdx++,
	frameVisibleBlocks.push(visibleBlockInfo);

	// This just returns the size of the pre-allocated buffer.
	long allocatedBlocks = scene->index.getNumAllocatedVoxelBlocks();
	// This is the number of blocks we are using out of the chunk that was allocated initially on
	// the GPU (for non-swapping case).
	long usedBlocks = allocatedBlocks - scene->localVBA.lastFreeBlockId;

	long allocatedExcessEntries = SDF_EXCESS_LIST_SIZE;
	long usedExcessEntries = allocatedExcessEntries - tempData->noAllocatedExcessEntries;

	if (usedBlocks > allocatedBlocks) {
		usedBlocks = allocatedBlocks;
	}
	if (usedExcessEntries > allocatedExcessEntries) {
		usedExcessEntries = allocatedExcessEntries;
	}

	// Display some memory stats, useful for debugging mapping failures.
	float percentFree = 100.0f * (1.0f - static_cast<float>(usedBlocks) / allocatedBlocks);
	float allocatedSizeMiB = scene->localVBA.allocatedSize * sizeof(ITMVoxel) / 1024.0f / 1024.0f;
	printf("[Visible: %6d | Used blocks (primary): %8ld/%ld (%.2f%% free)\n"
			" Used excess list slots: %8ld/%ld | Total allocated size: %.2fMiB]\n",
			tempData->noVisibleEntries,
			usedBlocks,
			allocatedBlocks,
			percentFree,
			usedExcessEntries,
			allocatedExcessEntries,
			allocatedSizeMiB);

	ITMHashEntry *entries = scene->index.GetEntries();
	if (scene->localVBA.lastFreeBlockId < 0) {
		fprintf(stderr, "ERROR: Last free block ID was negative (%d). This may indicate an "
				"allocation failure, causing your map to stop being able to grow.\n", scene->localVBA.lastFreeBlockId);
		throw std::runtime_error(
				"Invalid free voxel block ID. InfiniTAM has likely run out of GPU memory.");
	}
}

template<class TVoxel>
void ITMSceneReconstructionEngine_CUDA<TVoxel, ITMVoxelBlockHash>::IntegrateIntoScene(ITMScene<TVoxel, ITMVoxelBlockHash> *scene, const ITMView *view,
	const ITMTrackingState *trackingState, const ITMRenderState *renderState)
{
	Vector2i rgbImgSize = view->rgb->noDims;
	Vector2i depthImgSize = view->depth->noDims;
	float voxelSize = scene->sceneParams->voxelSize;

	Matrix4f M_d, M_rgb;
	Vector4f projParams_d, projParams_rgb;

	ITMRenderState_VH *renderState_vh = (ITMRenderState_VH*)renderState;
	if (renderState_vh->noVisibleEntries == 0) {
		// Our view has no useful data, so there's nothing to allocate. This happens, e.g., when
		// we fuse frames belonging to object instances, in which the actual instance is too far
		// away. Its depth values are over the max depth threshold (and, likely too noisy) and
		// they get ignored, leading to a blank ITMView with nothing new to integrate.
		return;
	}

	M_d = trackingState->pose_d->GetM();
	if (TVoxel::hasColorInformation) M_rgb = view->calib->trafo_rgb_to_depth.calib_inv * M_d;

	projParams_d = view->calib->intrinsics_d.projectionParamsSimple.all;
	projParams_rgb = view->calib->intrinsics_rgb.projectionParamsSimple.all;

	float mu = scene->sceneParams->mu; int maxW = scene->sceneParams->maxW;

	float *depth = view->depth->GetData(MEMORYDEVICE_CUDA);
	Vector4u *rgb = view->rgb->GetData(MEMORYDEVICE_CUDA);
	TVoxel *localVBA = scene->localVBA.GetVoxelBlocks();
	ITMHashEntry *hashTable = scene->index.GetEntries();

	int *visibleEntryIDs = renderState_vh->GetVisibleEntryIDs();

	dim3 cudaBlockSize(SDF_BLOCK_SIZE, SDF_BLOCK_SIZE, SDF_BLOCK_SIZE);
	dim3 gridSize(renderState_vh->noVisibleEntries);

	// These kernels are launched over ALL visible blocks, whose IDs are placed conveniently as the
	// first `renderState_vh->noVisibleEntries` elements of the `visibleEntryIDs` array, which could,
	// in theory, accommodate ALL possible blocks, but usually contains O(10k) blocks.
	if (scene->sceneParams->stopIntegratingAtMaxW) {
		if (trackingState->requiresFullRendering) {
			integrateIntoScene_device<TVoxel, true, false> << < gridSize, cudaBlockSize >> > (
				localVBA, hashTable, visibleEntryIDs, rgb, rgbImgSize, depth, depthImgSize,
				M_d, M_rgb, projParams_d, projParams_rgb, voxelSize, mu, maxW);
		} else {
			integrateIntoScene_device<TVoxel, true, true> << < gridSize, cudaBlockSize >> > (
				localVBA, hashTable, visibleEntryIDs, rgb, rgbImgSize, depth, depthImgSize,
				M_d, M_rgb, projParams_d, projParams_rgb, voxelSize, mu, maxW);
		}
	}
	else {
		if (trackingState->requiresFullRendering) {
			// While developing dynslam, this is the version that is run.
			integrateIntoScene_device<TVoxel, false, false> << < gridSize, cudaBlockSize >> > (
					localVBA, hashTable, visibleEntryIDs, rgb, rgbImgSize, depth, depthImgSize,
						M_d, M_rgb, projParams_d, projParams_rgb, voxelSize, mu, maxW);
		}
		else {
			integrateIntoScene_device<TVoxel, false, true> << < gridSize, cudaBlockSize >> > (
				localVBA, hashTable, visibleEntryIDs, rgb, rgbImgSize, depth, depthImgSize,
						M_d, M_rgb, projParams_d, projParams_rgb, voxelSize, mu, maxW);
		}
	}
}


template<class TVoxel>
int fullDecay(ITMScene<TVoxel, ITMVoxelBlockHash> *scene,
			   int maxWeight,
			   int *lastFreeBlockId_device
) {
	// TODO(andrei): Use custom block cleanup buffer, since we DO expect to do lots of work
	// here, and we're OK with the malloc overhead.
	fprintf(stderr, "WILL now decay ALL voxels in the map...\n");
  throw std::runtime_error("Don't use this now.");

	dim3 voxelBlockSize(SDF_BLOCK_SIZE, SDF_BLOCK_SIZE, SDF_BLOCK_SIZE);
	int *voxelAllocationList = scene->localVBA.GetAllocationList();
	TVoxel *localVBA = scene->localVBA.GetVoxelBlocks();
	ITMHashEntry *hashTable = scene->index.GetEntries();

	// TODO(andrei): Don't malloc anything in this method.
	// First, we check every bucket and see if it's allocated, populating each index
	// in `visibleBlockGlobalPos` with the block's position, whereby every element in
	// this array corresponds to a VBA element.
	long sdfLocalBlockNum = scene->index.getNumAllocatedVoxelBlocks();
	int noTotalEntries = scene->index.noTotalEntries;
	Vector4s *visibleBlockGlobalPos_device;
	ITMSafeCall(hipMalloc((void**)&visibleBlockGlobalPos_device, sdfLocalBlockNum * sizeof(Vector4s)));
	ITMSafeCall(hipMemset(visibleBlockGlobalPos_device, 0, sizeof(Vector4s) * sdfLocalBlockNum));

	dim3 hashTableVisitBlockSize(1024);
	dim3 hashTableVisitGridSize((noTotalEntries - 1) / hashTableVisitBlockSize.x + 1);

	fprintf(stderr, "Launching findAllocatedBlocks with gs.x = %d\n", hashTableVisitGridSize.x);
	fprintf(stderr, "total entries: %d %x\n", noTotalEntries, noTotalEntries);
	ITMLib::Engine::findAllocatedBlocks<<<hashTableVisitGridSize, hashTableVisitBlockSize>>>(
			visibleBlockGlobalPos_device, hashTable, noTotalEntries
	);
	ITMSafeCall(hipDeviceSynchronize());
	ITMSafeCall(hipGetLastError());
	printf("Aight, computed allocated blocks OK...\n");

	// We now know, for every block allocated in the VBA, whether it's in use, and what its
	// global coordinates are.
	dim3 gridSize(sdfLocalBlockNum);
	fprintf(stderr, "Calling decayFull_device...: gs.x = %d\n", gridSize.x);

	int oldLastFreeBlockId = scene->localVBA.lastFreeBlockId;
	decayFull_device<TVoxel> <<< gridSize, voxelBlockSize >>> (
			visibleBlockGlobalPos_device,
					localVBA,
					hashTable,
					maxWeight,
					lastFreeBlockId_device,
					voxelAllocationList);
	ITMSafeCall(hipDeviceSynchronize());
	ITMSafeCall(hipGetLastError());
	printf("decayFull_device went OK\n\n");

	ITMSafeCall(hipMemcpy(&(scene->localVBA.lastFreeBlockId), lastFreeBlockId_device,
				1 * sizeof(int),
				hipMemcpyDeviceToHost));
	int freedBlockCount = scene->localVBA.lastFreeBlockId - oldLastFreeBlockId;

	printf("decayFull_device deleted %d blocks\n\n", freedBlockCount);

	ITMSafeCall(hipFree(visibleBlockGlobalPos_device));
	return freedBlockCount;
}

template<class TVoxel>
void ITMSceneReconstructionEngine_CUDA<TVoxel, ITMVoxelBlockHash>::Decay(
		ITMScene<TVoxel, ITMVoxelBlockHash> *scene,
		int maxWeight,
		int minAge,
		bool forceAllVoxels
) {
	// TODO(andrei): Refactor this method once the functionality is more or less complete.

//	const bool deallocateEmptyBlocks = false;	// This could be a config param of the recon engine.
	int oldLastFreeBlockId = scene->localVBA.lastFreeBlockId;

	int *voxelAllocationList = scene->localVBA.GetAllocationList();
	TVoxel *localVBA = scene->localVBA.GetVoxelBlocks();
	ITMHashEntry *hashTable = scene->index.GetEntries();

	ITMSafeCall(hipMemcpy(lastFreeBlockId_device, &(scene->localVBA.lastFreeBlockId),
						   1 * sizeof(int),
						   hipMemcpyHostToDevice));

	dim3 voxelBlockSize(SDF_BLOCK_SIZE, SDF_BLOCK_SIZE, SDF_BLOCK_SIZE);

	if (forceAllVoxels) {
		// TODO(andrei): Make this function obey the 'deallocateEmptryBlocks' flag.
		// TODO(andrei): Remove duplicate functionality for counting freed blocks.
		fullDecay<TVoxel>(scene, maxWeight, this->lastFreeBlockId_device);
	}
	else if (frameVisibleBlocks.size() > minAge) {
		VisibleBlockInfo visible = frameVisibleBlocks.front();
		frameVisibleBlocks.pop();

		printf("Running decay_device on the %lu blocks visible at frame %lu.\n",
			   visible.count,
			   visible.frameIdx);

		// Ensure there are voxels to work with. We can often encounter empty frames when
		// reconstructing individual objects which are too far from the camera for any
		// meaningful depth to be estimated, so there's nothing to do for them.
		if (visible.count > 0) {
			ITMSafeCall(hipMemset( locks_device, 0, SDF_BUCKET_NUM * sizeof(int)));

			dim3 gridSize(static_cast<uint32_t>(visible.count));
			decay_device<TVoxel> <<< gridSize, voxelBlockSize >>> (
					localVBA,
					hashTable,
					visible.blockIDs->GetData(MEMORYDEVICE_CUDA),
					minAge,
					maxWeight,
					voxelAllocationList,
					lastFreeBlockId_device,
					locks_device,
					frameIdx
			);
			ITMSafeCall(hipDeviceSynchronize());
			ITMSafeCall(hipGetLastError());

			// This is important for ensuring ITM "knows" about the freed up blocks in the VBA.
			ITMSafeCall(hipMemcpy(&(scene->localVBA.lastFreeBlockId), lastFreeBlockId_device,
								   1 * sizeof(int),
								   hipMemcpyDeviceToHost));

			delete visible.blockIDs;
		}
	}

	int freedBlockCount = scene->localVBA.lastFreeBlockId - oldLastFreeBlockId;
	totalDecayedBlockCount += freedBlockCount;

	// TODO(andrei): New benchmarks once the complete implementation is in place!
	// Note: no explicit cleanup was done at the end of the sequences!
	// Mini-bench: 50 frames starting with 3900 in odometry sequence 08.
	// sdfLocalBlockNum: 0x80000
	// With 			ELAS no pruning: 90.50% free
	// With (w=1, a=3)  ELAS pruning:    95.35% free
	// With (w=3, a=10) ELAS pruning:    95.57% free
	// With (w=5, a=10) ELAS extreme:    96.92% free
	//
	// 75 frames now, 0x40000 (old InfiniTAM default)
	// With 			dispnet no pruning: 51.04% free
	// With (w=1, a=3)  dispnet pruning:    68.89% free
	// With (w=2, a=10) dispnet pruning:    71.40% free (visually this seems the best)
	// With (w=3, a=10) dispnet pruning:    75.63% free (a little harsh, but looks OK)
	// With (w=5, a=15) dispnet extreme:    79.22% free (extreme)

	if (freedBlockCount > 0) {
		size_t savings = sizeof(TVoxel) * SDF_BLOCK_SIZE3 * freedBlockCount;
		float savingsMb = (savings / 1024.0f / 1024.0f);

		printf("Found %d candidate blocks to deallocate with weight [%d] or below and age [%d]. "
			   "Saved %.2fMb.\n",
			   freedBlockCount,
			   maxWeight,
			   minAge,
			   savingsMb);
	}
	else {
		printf("Decay process found NO voxel blocks to deallocate.\n");
	}
}


template<class TVoxel>
size_t ITMSceneReconstructionEngine_CUDA<TVoxel, ITMVoxelBlockHash>::GetDecayedBlockCount() {
	return static_cast<size_t>(totalDecayedBlockCount);
}

// plain voxel array

template<class TVoxel>
void ITMSceneReconstructionEngine_CUDA<TVoxel,ITMPlainVoxelArray>::ResetScene(ITMScene<TVoxel, ITMPlainVoxelArray> *scene)
{
	int numBlocks = scene->index.getNumAllocatedVoxelBlocks();
	int blockSize = scene->index.getVoxelBlockSize();

	TVoxel *voxelBlocks_ptr = scene->localVBA.GetVoxelBlocks();
	memsetKernel<TVoxel>(voxelBlocks_ptr, TVoxel(), numBlocks * blockSize);
	int *vbaAllocationList_ptr = scene->localVBA.GetAllocationList();
	fillArrayKernel<int>(vbaAllocationList_ptr, numBlocks);
	scene->localVBA.lastFreeBlockId = numBlocks - 1;
}

template<class TVoxel>
void ITMSceneReconstructionEngine_CUDA<TVoxel, ITMPlainVoxelArray>::AllocateSceneFromDepth(ITMScene<TVoxel, ITMPlainVoxelArray> *scene, const ITMView *view,
	const ITMTrackingState *trackingState, const ITMRenderState *renderState, bool onlyUpdateVisibleList)
{
}

template<class TVoxel>
void ITMSceneReconstructionEngine_CUDA<TVoxel, ITMPlainVoxelArray>::IntegrateIntoScene(ITMScene<TVoxel, ITMPlainVoxelArray> *scene, const ITMView *view,
	const ITMTrackingState *trackingState, const ITMRenderState *renderState)
{
	Vector2i rgbImgSize = view->rgb->noDims;
	Vector2i depthImgSize = view->depth->noDims;
	float voxelSize = scene->sceneParams->voxelSize;

	Matrix4f M_d, M_rgb;
	Vector4f projParams_d, projParams_rgb;

	M_d = trackingState->pose_d->GetM();
	if (TVoxel::hasColorInformation) M_rgb = view->calib->trafo_rgb_to_depth.calib_inv * M_d;

	projParams_d = view->calib->intrinsics_d.projectionParamsSimple.all;
	projParams_rgb = view->calib->intrinsics_rgb.projectionParamsSimple.all;

	float mu = scene->sceneParams->mu; int maxW = scene->sceneParams->maxW;

	float *depth = view->depth->GetData(MEMORYDEVICE_CUDA);
	Vector4u *rgb = view->rgb->GetData(MEMORYDEVICE_CUDA);
	TVoxel *localVBA = scene->localVBA.GetVoxelBlocks();
	const ITMPlainVoxelArray::ITMVoxelArrayInfo *arrayInfo = scene->index.getIndexData();

	dim3 cudaBlockSize(8, 8, 8);
	dim3 gridSize(
		scene->index.getVolumeSize().x / cudaBlockSize.x,
		scene->index.getVolumeSize().y / cudaBlockSize.y,
		scene->index.getVolumeSize().z / cudaBlockSize.z);

	if (scene->sceneParams->stopIntegratingAtMaxW) {
		if (trackingState->requiresFullRendering)
			integrateIntoScene_device < TVoxel, true, false> << <gridSize, cudaBlockSize >> >(localVBA, arrayInfo,
				rgb, rgbImgSize, depth, depthImgSize, M_d, M_rgb, projParams_d, projParams_rgb, voxelSize, mu, maxW);
		else
			integrateIntoScene_device < TVoxel, true, true> << <gridSize, cudaBlockSize >> >(localVBA, arrayInfo,
				rgb, rgbImgSize, depth, depthImgSize, M_d, M_rgb, projParams_d, projParams_rgb, voxelSize, mu, maxW);
	}
	else
	{
		if (trackingState->requiresFullRendering)
			integrateIntoScene_device < TVoxel, false, false> << <gridSize, cudaBlockSize >> >(localVBA, arrayInfo,
				rgb, rgbImgSize, depth, depthImgSize, M_d, M_rgb, projParams_d, projParams_rgb, voxelSize, mu, maxW);
		else
			integrateIntoScene_device < TVoxel, false, true> << <gridSize, cudaBlockSize >> >(localVBA, arrayInfo,
				rgb, rgbImgSize, depth, depthImgSize, M_d, M_rgb, projParams_d, projParams_rgb, voxelSize, mu, maxW);
	}
}

template<class TVoxel>
void ITMSceneReconstructionEngine_CUDA<TVoxel, ITMPlainVoxelArray>::Decay(
		ITMScene<TVoxel, ITMPlainVoxelArray>*, int, int, bool
) {
  throw std::runtime_error("Map decay is not supported in conjunction with plain voxel arrays, "
						   "only with voxel block hashing.");
}

template<class TVoxel>
size_t ITMSceneReconstructionEngine_CUDA<TVoxel, ITMPlainVoxelArray>::GetDecayedBlockCount() {
	throw std::runtime_error("Map decay is not supported in conjunction with plain voxel arrays, "
							 "only with voxel block hashing.");
}


// device functions

template<class TVoxel, bool stopMaxW, bool approximateIntegration>
__global__ void integrateIntoScene_device(TVoxel *voxelArray, const ITMPlainVoxelArray::ITMVoxelArrayInfo *arrayInfo,
	const Vector4u *rgb, Vector2i rgbImgSize, const float *depth, Vector2i depthImgSize, Matrix4f M_d, Matrix4f M_rgb, Vector4f projParams_d, 
	Vector4f projParams_rgb, float _voxelSize, float mu, int maxW)
{
	int x = blockIdx.x*blockDim.x+threadIdx.x;
	int y = blockIdx.y*blockDim.y+threadIdx.y;
	int z = blockIdx.z*blockDim.z+threadIdx.z;

	Vector4f pt_model; int locId;

	locId = x + y * arrayInfo->size.x + z * arrayInfo->size.x * arrayInfo->size.y;
	
	if (stopMaxW) if (voxelArray[locId].w_depth == maxW) return;
//	if (approximateIntegration) if (voxelArray[locId].w_depth != 0) return;

	pt_model.x = (float)(x + arrayInfo->offset.x) * _voxelSize;
	pt_model.y = (float)(y + arrayInfo->offset.y) * _voxelSize;
	pt_model.z = (float)(z + arrayInfo->offset.z) * _voxelSize;
	pt_model.w = 1.0f;

	ComputeUpdatedVoxelInfo<TVoxel::hasColorInformation,TVoxel>::compute(
			voxelArray[locId],
			pt_model,
			M_d,
			projParams_d,
			M_rgb,
			projParams_rgb,
			mu,
			maxW,
			depth,
			depthImgSize,
			rgb,
			rgbImgSize);
}

template<class TVoxel, bool stopMaxW, bool approximateIntegration>
__global__ void integrateIntoScene_device(TVoxel *localVBA,
										  const ITMHashEntry *hashTable,
										  int *visibleEntryIDs,
										  const Vector4u *rgb,
										  Vector2i rgbImgSize,
										  const float *depth, Vector2i depthImgSize,
										  Matrix4f M_d, Matrix4f M_rgb,
										  Vector4f projParams_d,
										  Vector4f projParams_rgb,
										  float _voxelSize,
										  float mu,
										  int maxW)
{
	Vector3i globalPos;
	int entryId = visibleEntryIDs[blockIdx.x];

	/// XXX: should we re-look it up in the hash table just to be sure?
	const ITMHashEntry &currentHashEntry = hashTable[entryId];

	// What error message could we show here for mistakes in integration?

	if (currentHashEntry.ptr < 0) return;

	globalPos = currentHashEntry.pos.toInt() * SDF_BLOCK_SIZE;

	TVoxel *localVoxelBlock = &(localVBA[currentHashEntry.ptr * SDF_BLOCK_SIZE3]);

	int x = threadIdx.x, y = threadIdx.y, z = threadIdx.z;

	Vector4f pt_model; int locId;

	locId = x + y * SDF_BLOCK_SIZE + z * SDF_BLOCK_SIZE * SDF_BLOCK_SIZE;

	if (stopMaxW) if (localVoxelBlock[locId].w_depth == maxW) return;
	if (approximateIntegration) if (localVoxelBlock[locId].w_depth != 0) return;

	pt_model.x = (float)(globalPos.x + x) * _voxelSize;
	pt_model.y = (float)(globalPos.y + y) * _voxelSize;
	pt_model.z = (float)(globalPos.z + z) * _voxelSize;
	pt_model.w = 1.0f;

	ComputeUpdatedVoxelInfo<TVoxel::hasColorInformation,TVoxel>::compute(localVoxelBlock[locId], pt_model, M_d, projParams_d, M_rgb, projParams_rgb, mu, maxW, depth, depthImgSize, rgb, rgbImgSize);
}

__global__ void buildHashAllocAndVisibleType_device(
		uchar *entriesAllocType,
		uchar *entriesVisibleType,
		Vector4s *blockCoords,
		const float *depth,
		Matrix4f invM_d,
		Vector4f projParams_d,
		float mu,
		Vector2i _imgSize,
		float _voxelSize,
		ITMHashEntry *hashTable,
		float viewFrustum_min,
		float viewFrustum_max,
		int *locks
) {
	int x = threadIdx.x + blockIdx.x * blockDim.x, y = threadIdx.y + blockIdx.y * blockDim.y;

	if (x > _imgSize.x - 1 || y > _imgSize.y - 1) return;

	buildHashAllocAndVisibleTypePP(entriesAllocType, entriesVisibleType, x, y, blockCoords, depth, invM_d,
		projParams_d, mu, _imgSize, _voxelSize, hashTable, viewFrustum_min, viewFrustum_max, locks);
}

__global__ void setToType3(uchar *entriesVisibleType, int *visibleEntryIDs, int noVisibleEntries,
						   ITMHashEntry *hashTable)
{
	int entryId = threadIdx.x + blockIdx.x * blockDim.x;
	if (entryId > noVisibleEntries - 1) return;

	if (hashTable[visibleEntryIDs[entryId]].ptr < -1) {
		if (entryId % 100 == 17) {
			printf("Was trying to set entry ID #%d as visible type 3 but it had been deallocated!\n",
				   entryId);
			// TODO should we make it 0 or something? Or type #4 == recycled?
		}

		/// XXX: is this sensible?
//		entriesVisibleType[visibleEntryIDs[entryId]] = 42;
	}
	else {
		entriesVisibleType[visibleEntryIDs[entryId]] = 3;
	}
}

__global__ void allocateVoxelBlocksList_device(
		int *voxelAllocationList, int *excessAllocationList,
		ITMHashEntry *hashTable, int noTotalEntries,
		AllocationTempData *allocData,
		uchar *entriesAllocType, uchar *entriesVisibleType,
		Vector4s *blockCoords,
		int currentFrame
) {
	int targetIdx = threadIdx.x + blockIdx.x * blockDim.x;
	if (targetIdx > noTotalEntries - 1) return;

	int vbaIdx, exlIdx;

	switch (entriesAllocType[targetIdx])
	{
		case 0: // TODO(andrei): Could we please use constants/enums/defines for these values?
			// 0 == Invisible block.
		break;

	case 1:
		// 1 == block visible and needs allocation, fits in the ordered list.
		vbaIdx = atomicSub(&allocData->noAllocatedVoxelEntries, 1);

		if (vbaIdx >= 0) //there is room in the voxel block array
		{
			Vector4s pt_block_all = blockCoords[targetIdx];

			ITMHashEntry hashEntry;
			hashEntry.pos.x = pt_block_all.x; hashEntry.pos.y = pt_block_all.y; hashEntry.pos.z = pt_block_all.z;
			hashEntry.ptr = voxelAllocationList[vbaIdx];
			hashEntry.offset = 0;
			hashEntry.allocatedTime = currentFrame;

			// TODO(andrei): What if there are multiple blocks which think they belong in the
			// ordered list?
			hashTable[targetIdx] = hashEntry;
		}
		else
		{
			// TODO(andrei): Handle this better.
			printf("WARNING: No more room in VBA! vbaIdx became %d.\n", vbaIdx);
			printf("exlIdx is %d.\n", allocData->noAllocatedExcessEntries);
		}
		break;

	case 2:
		// 2 == block visible and needs allocation in the excess list
		vbaIdx = atomicSub(&allocData->noAllocatedVoxelEntries, 1);
		exlIdx = atomicSub(&allocData->noAllocatedExcessEntries, 1);

		if (vbaIdx >= 0 && exlIdx >= 0) //there is room in the voxel block array and excess list
		{
			Vector4s pt_block_all = blockCoords[targetIdx];

			ITMHashEntry hashEntry;
			hashEntry.pos.x = pt_block_all.x; hashEntry.pos.y = pt_block_all.y; hashEntry.pos.z = pt_block_all.z;
			hashEntry.ptr = voxelAllocationList[vbaIdx];
			hashEntry.offset = 0;
			hashEntry.allocatedTime = currentFrame;

			int exlOffset = excessAllocationList[exlIdx];

			hashTable[targetIdx].offset = exlOffset + 1; //connect to child

			hashTable[SDF_BUCKET_NUM + exlOffset] = hashEntry; //add child to the excess list

			entriesVisibleType[SDF_BUCKET_NUM + exlOffset] = 1; //make child visible
		}
		else
		{
			// TODO(andrei): Handle this better. We could probably get away with just looking at
			// noAllocatedVoxelEntries and noAllocatedExcessEntries after the kernel completes.
			if (vbaIdx >= 0)
			{
				printf("WARNING: Could not allocate in excess list! There was still room in the main VBA, "
						   "but exlIdx = %d! Consider increasing the overall hash table size, or at least the "
						   "bucket size.\n", exlIdx);
			}
			else if(exlIdx)
			{
				printf("WARNING: Tried to allocate in excess list, but failed because the main VBA is "
							 "full. vbaIdx = %d\n", vbaIdx);
			}
			else
			{
				printf("WARNING: No more room in VBA or in the excess list! vbaIdx became %d.\n", vbaIdx);
				printf("exlIdx is %d.\n", exlIdx);
			}
		}
		break;

		default:
			printf("Unexpected alloc type: %d\n", static_cast<int>(entriesAllocType[targetIdx]));
		break;
	}
}

__global__ void reAllocateSwappedOutVoxelBlocks_device(int *voxelAllocationList, ITMHashEntry *hashTable, int noTotalEntries,
	AllocationTempData *allocData, /*int *noAllocatedVoxelEntries,*/ uchar *entriesVisibleType)
{
	int targetIdx = threadIdx.x + blockIdx.x * blockDim.x;
	if (targetIdx > noTotalEntries - 1) return;

	int vbaIdx;
	int hashEntry_ptr = hashTable[targetIdx].ptr;

	if (entriesVisibleType[targetIdx] > 0 && hashEntry_ptr == -1) //it is visible and has been previously allocated inside the hash, but deallocated from VBA
	{
		vbaIdx = atomicSub(&allocData->noAllocatedVoxelEntries, 1);
		if (vbaIdx >= 0) hashTable[targetIdx].ptr = voxelAllocationList[vbaIdx];
	}
}

template<bool useSwapping>
__global__ void buildVisibleList_device(
		ITMHashEntry *hashTable,
		ITMHashSwapState *swapStates,
		int noTotalEntries,
		int *visibleEntryIDs,
		AllocationTempData *allocData,
		uchar *entriesVisibleType,
		Matrix4f M_d,
		Vector4f projParams_d,
		Vector2i depthImgSize,
		float voxelSize
) {
	int targetIdx = threadIdx.x + blockIdx.x * blockDim.x;
	if (targetIdx > noTotalEntries - 1) return;

	__shared__ bool shouldPrefix;
	shouldPrefix = false;
	__syncthreads();

	unsigned char hashVisibleType = entriesVisibleType[targetIdx];
	const ITMHashEntry & hashEntry = hashTable[targetIdx];

	// i.e., previously seen
	if (hashVisibleType == 3)
	{
		bool isVisibleEnlarged, isVisible;

		if (useSwapping)
		{
			checkBlockVisibility<true>(isVisible, isVisibleEnlarged, hashEntry.pos, M_d, projParams_d, voxelSize, depthImgSize);
			if (!isVisibleEnlarged) hashVisibleType = 0;
		} else {
			checkBlockVisibility<false>(isVisible, isVisibleEnlarged, hashEntry.pos, M_d, projParams_d, voxelSize, depthImgSize);
			if (!isVisible) hashVisibleType = 0;
		}
		entriesVisibleType[targetIdx] = hashVisibleType;
	}

	if (hashVisibleType > 0) shouldPrefix = true;

	if (useSwapping)
	{
		if (hashVisibleType > 0 && swapStates[targetIdx].state != 2) swapStates[targetIdx].state = 1;
	}

	__syncthreads();

	// Computes the correct offsets for the visible blocks in `visibleEntryIDs`.
	if (shouldPrefix)
	{
		int offset = computePrefixSum_device<int>(hashVisibleType > 0,
												  &allocData->noVisibleEntries,
												  blockDim.x * blockDim.y,
												  threadIdx.x);
		if (offset != -1) visibleEntryIDs[offset] = targetIdx;
	}

#if 0
	// "active list": blocks that have new information from depth image
	// currently not used...
	__syncthreads();

	if (shouldPrefix)
	{
		int offset = computePrefixSum_device<int>(hashVisibleType == 1, noActiveEntries, blockDim.x * blockDim.y, threadIdx.x);
		if (offset != -1) activeEntryIDs[offset] = targetIdx;
	}
#endif
}

// TODO(andrei): Get rid of code duplication.

/// \brief Deletes a block from the hash table, deallocating its VBA entry.
/// \param hashTable
/// \param blockPos             The position of the block in the voxel grid, i.e., the key.
/// \param locks                Array used for locking in order to prevent data races when
///                             attempting to delete multiple elements with the same key.
/// \param lastFreeBlockId      Index in the voxel allocation list (free list).
/// \param voxelAllocationList  List of free voxels.
///
/// \note Does not support swapping.
template<class TVoxel>
__device__
void deleteBlock(
		ITMHashEntry *hashTable,
		Vector3i blockPos,
		int *locks,
		int *voxelAllocationList,
		int *lastFreeBlockId
) {
	int keyHash = hashIndex(blockPos);
	int contention = atomicAdd(&locks[keyHash], 1);
	if (contention > 0) {
		printf("Contention on bucket of hash value %d. Not going further with deletion of block "
					   "(%d, %d, %d).\n", keyHash, blockPos.x, blockPos.y, blockPos.z);
		atomicSub(&locks[keyHash], 1);
		return;
	}

	bool isFound = false;
	int outBlockIdx = -1;
	int outPrevBlockIdx = -1;
	findVoxel(hashTable, blockPos, 0, isFound, outBlockIdx, outPrevBlockIdx);

	bool isExcess = (outBlockIdx >= SDF_BUCKET_NUM);
	// Paranoid sanity check
	if (outPrevBlockIdx == -1) {
		if (isExcess) {
			printf("\n[ERROR] Found entity in excess list with no previous element (%d, %d, %d)!\n",
				   blockPos.x,
				   blockPos.y,
				   blockPos.z);
		}
	}
	else {
		if (! isExcess) {
			printf("\n[ERROR] Found entity in bucket list with a previous guy!\n");
		}
	}

	bool hasNext = (hashTable[outBlockIdx].offset >= 1);
	if (isExcess || hasNext) {
		// Even after tons of careful debugging, there still seem to be a few Heisenbugs around
		// causing the wrong blocks to disappear when deleting an element from a bucket with more
		// than one element. However, if our hash table's size is large enough, this happens quite
		// rarely, so we ignore it for now, since we save enough memory by deleting single-block
		// buckets anyway.
		// TODO(andrei): See if you can isolate the issue to ONLY 'isExcess' or 'hasNext'.
		atomicSub(&locks[keyHash], 1);
		return;
	}

	// Some paranoid sanity checks. TODO(andrei): Consider adding flag for toggling.
	if (!isFound || outBlockIdx < 0) {
//		if (blockPos.x % 10 == 3) {
			printf("\n\nFATAL ERROR: sanity check failed in 'decay_device' voxel (block) "
						   "found = %d, outBlockIdx = %d (%d, %d, %d) ; %s.\n",
				   static_cast<int>(isFound),
				   outBlockIdx,
				   blockPos.x,
				   blockPos.y,
				   blockPos.z,
				   isExcess ? "excess" : "non-excess"
			);
//		}
		atomicSub(&locks[keyHash], 1);
		return;
	}

//	if (blockPos.x % 100 == 17) {
//		printf("Will delete block with hash idx %d (prev = %d); (%d, %d, %d); %s\n",
//			   outBlockIdx,
//			   outPrevBlockIdx,
//			   blockPos.x,
//			   blockPos.y,
//			   blockPos.z,
//			   isExcess ? "excess" : "non-excess"
//		);
//	}

	// First, deallocate the VBA slot.
	int freeListIdx = atomicAdd(&lastFreeBlockId[0], 1);
	voxelAllocationList[freeListIdx] = hashTable[outBlockIdx].ptr;

	// Finally, do bookkeeping for buckets with more than one element.
	// TODO(andrei): Update excess freelist! (Should work without doing it but leak memory.)

	if (outPrevBlockIdx != -1) {
		// In excess list with a successor or not.
		hashTable[outPrevBlockIdx].offset = hashTable[outBlockIdx].offset;
		hashTable[outBlockIdx].offset = 0;
		hashTable[outBlockIdx].ptr = -2;
	}
	else {
		if (hashTable[outBlockIdx].offset >= 1) {
			// In ordered list, with a successor.
			long nextIdx = SDF_BUCKET_NUM + hashTable[outBlockIdx].offset - 1;
			// Note: this invalidates the visible ID list by moving stuff around.
			hashTable[outBlockIdx] = hashTable[nextIdx];

			// Free up the slot we just copied into the main VBA, in case there's still pointers
			// to it in the visible list from some to-be-decayed frame.
			// [RIP] Not doing this can mean the zombie block gets detected as valid in the future,
			// even though it's in the excess area but nobody is pointing at it.
			hashTable[nextIdx].offset = 0;
			hashTable[nextIdx].ptr = -2;
		}
		else {
			// In ordered list, and no successor.
			hashTable[outBlockIdx].offset = 0;
			hashTable[outBlockIdx].ptr = -2;
		}
	}

	// Release the lock.
	atomicSub(&locks[keyHash], 1);
}

// This kernel runs per-voxel, just like 'decayFull_device'.
template<class TVoxel>
__global__
void decay_device(TVoxel *localVBA,
				  ITMHashEntry *hashTable,
				  int *visibleEntryIDs,
				  int minAge,
				  int maxWeight,
				  int *voxelAllocationList,
				  int *lastFreeBlockId,
				  int *locks,
				  int currentFrame
) {
	// Note: there are no range checks because we launch exactly as many threads as we need.
	int entryId = visibleEntryIDs[blockIdx.x];

	// Possibly now points to the wrong place. We just take out the position, and re-look it up.
	const ITMHashEntry &currentHashEntry = hashTable[entryId];
	static const int voxelsPerBlock = SDF_BLOCK_SIZE3;

	// The local offset of the voxel in the current block.
	int locId = threadIdx.x + threadIdx.y * SDF_BLOCK_SIZE + threadIdx.z * SDF_BLOCK_SIZE * SDF_BLOCK_SIZE;

	Vector3i intPos = currentHashEntry.pos.toInt();
	// The global position of the voxel block.
	Vector3i globalPos = intPos * SDF_BLOCK_SIZE;
	bool isFound = false;
	int blockHashIdx = -1;
	int blockPrevHashIdx = -1;
	int voxelIdx = findVoxel(hashTable, intPos, locId, isFound, blockHashIdx, blockPrevHashIdx);

	if (-1 == blockHashIdx) {
		// This happens when we, for instance, have an ID in the visible list whose target gets
		// deleted from the hash table by a previous decay phase.
//		if (locId == 0) {
//			int hashVal = hashIndex(intPos);
//			printf("ERROR: could not find bucket for (%d, %d, %d) @ hash ID %d.\n",
//                   intPos.x, intPos.y, intPos.z, hashVal);
//		}
		return;
	}

	if (! isFound && locId == 0 && blockIdx.x % 100 == 3) {
		printf("ERROR: voxel not found? WTF.\n");
		return;
	}

	bool emptyVoxel = false;
	bool safeToClear = true;
	int age = currentFrame - hashTable[blockHashIdx].allocatedTime;
	if (age < minAge) {
		// Important corner case: when we had a block in the visible list, but it got deleted in
		// a previous decay pass, and ended up also getting reallocated (and thus the old ID in
		// the visible list was pointing to the wrong thing).
		safeToClear = false;
	}

	if (safeToClear) {
		if (localVBA[voxelIdx].w_depth <= maxWeight && localVBA[voxelIdx].w_depth > 0) {
			localVBA[voxelIdx].reset();
			emptyVoxel = true;
		}

		if (localVBA[voxelIdx].w_depth == 0) {
			emptyVoxel = true;
		}
	}

	// TODO(andrei): Try summing all the weights and empty == weightSum < k (==3-10). Niessner et
	// al. do this.
	// Prepare for counting the number of empty voxels in each block.
	__shared__ int countBuffer[voxelsPerBlock];
	countBuffer[locId] = static_cast<int>(emptyVoxel);
	__syncthreads();

	// Block-level sum for counting non-empty voxels in this block.
	blockReduce(countBuffer, voxelsPerBlock, locId);
	__syncthreads();

	int emptyVoxels = countBuffer[0];
	bool emptyBlock = (emptyVoxels == voxelsPerBlock);

	// If we decide to clear up the block, let's be REALLY sure it's blank.
	if (emptyBlock) {
		if (localVBA[voxelIdx].w_depth != 0) {
			printf("SANITY CHECK ERROR FAIL: depth weight != 0\n");
		}
		if (localVBA[voxelIdx].sdf != localVBA[voxelIdx].SDF_initialValue()) {
			printf("SANITY CHECK FAIL: sdf nonzero somewhere in this here block..\n");
		}
		localVBA[voxelIdx].reset();
	}
	__syncthreads();

	if (locId == 0 && emptyBlock && safeToClear) {
		// Not recycling memory at the moment due to bugs.
//		deleteBlock<TVoxel>(hashTable,
//							currentHashEntry.pos.toInt(),
//							locks,
//							voxelAllocationList,
//							lastFreeBlockId);
	}
}


template<class TVoxel>
__global__
void decayFull_device(
		const Vector4s *visibleBlockGlobalPos,
		TVoxel *localVBA,
		ITMHashEntry *hashTable,
		int maxWeight,
		int *lastFreeBlockId,
		int *voxelAllocationList
) {
	const int voxelBlockIdx = blockIdx.x;
	const Vector4s blockGridPos_4s = visibleBlockGlobalPos[voxelBlockIdx];

	if (blockGridPos_4s.w == 0) {
		// A zero means no hash table entry points to this block.
		return;
	}

	const Vector3i localVoxPos(threadIdx.x, threadIdx.y, threadIdx.z);
	// Note: this also acts as the key for the voxel block hash.
	const Vector3i blockGridPos = Vector3i(blockGridPos_4s.x, blockGridPos_4s.y, blockGridPos_4s.z);
	const Vector3i blockPos = blockGridPos * SDF_BLOCK_SIZE;
	const Vector3i globalVoxPos = blockPos + localVoxPos;
	int locId = threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.y * blockDim.x;

	bool isFound = false;
	// Note: since we're operating on allocated blocks exclusively, then we must always FIND the
	// voxel. TODO(andrei): Should we assert that here?
	int blockHashIdx = -1;
	int blockPrevHashIdx = -1;
	int voxelIdx = findVoxel(hashTable, blockGridPos, locId, isFound, blockHashIdx, blockPrevHashIdx);

	if (-1 == blockHashIdx) {
		printf("ERROR: could not find bucket.\n");
		return;
	}

	if (! isFound && locId == 0 && blockIdx.x % 1000 == 3) {
		printf("ERROR: voxel not found? WTF.\n");
		return;
	}

	// TODO(andrei): Unconstantify weight threshold.
	bool emptyVoxel = false;
	if (localVBA[voxelIdx].w_depth <= 3 && localVBA[voxelIdx].w_depth > 0) {
//		if (localVoxPos.x == 0 && localVoxPos.y == 0 && localVoxPos.z == 0) {
//		}
		localVBA[voxelIdx].reset();
		emptyVoxel = true;
	}

	if (localVBA[voxelIdx].w_depth == 0) {
		emptyVoxel = true;
	}

	const int voxelsPerBlock = SDF_BLOCK_SIZE3;

	// Prepare for counting the number of empty voxels in each block.
	__shared__ int countBuffer[voxelsPerBlock];
	countBuffer[locId] = static_cast<int>(emptyVoxel);
	__syncthreads();

	// Block-level sum for counting non-empty voxels in this block.
	blockReduce(countBuffer, voxelsPerBlock, locId);
	__syncthreads();
	int emptyVoxelCount = countBuffer[0];
	// TODO(andrei): Use the tips from the original Niessner paper for GC: Min-SDF and max-weight thresholds.

	// TODO(andrei): Re-add once deleting is sync'd properly.
//	bool blockEmpty = (emptyVoxelCount == voxelsPerBlock);
//	if (locId == 0 && blockEmpty) {
//		deleteEntry<TVoxel>(hashTable, blockHashIdx, blockPrevHashIdx, lastFreeBlockId, voxelAllocationList);
//	}
}


template class ITMLib::Engine::ITMSceneReconstructionEngine_CUDA<ITMVoxel, ITMVoxelIndex>;

